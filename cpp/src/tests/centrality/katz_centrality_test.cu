#include "gtest/gtest.h"
#include "gmock/gmock.h"
#include "gmock/gmock-generated-matchers.h"
#include "high_res_clock.h"
#include "hip/hip_runtime_api.h"
#include <cugraph.h>
#include "test_utils.h"
#include <thrust/device_ptr.h>
#include <fstream>

std::vector<int>
getGoldenTopKIds(std::ifstream& fs_result, int k = 10) {
  std::vector<int> vec;
  int val;
  int count = 0;
  while (fs_result>>val && ((count++) < k)) {
    vec.push_back(val);
  }
  vec.resize(k);
  return vec;
}

std::vector<int>
getTopKIds(gdf_column_ptr katz, int k = 10) {
  int count = katz.get()->size;
  hipStream_t stream = nullptr;
  rmm::device_vector<int> id(count);
  thrust::sequence(rmm::exec_policy(stream)->on(stream), id.begin(), id.end());
  auto colptr = thrust::device_pointer_cast(static_cast<double*>(katz.get()->data));
  thrust::sort_by_key(rmm::exec_policy(stream)->on(stream),
      colptr, colptr + count, id.begin(), thrust::greater<double>());
  std::vector<int> topK(k);
  thrust::copy(id.begin(), id.begin() + k, topK.begin());
  return topK;
}

int
getMaxDegree(cugraph::Graph * G) {
      cugraph::add_adj_list(G);
      std::vector<int> out_degree(G->numberOfVertices);
      gdf_column_ptr col_out_degree = create_gdf_column(out_degree);
      cugraph::degree(G, col_out_degree.get(), 2);
      auto degreePtr = thrust::device_pointer_cast(static_cast<int*>(col_out_degree.get()->data));
      hipStream_t stream = nullptr;
      int max_out_degree = thrust::reduce(rmm::exec_policy(stream)->on(stream),
          degreePtr, degreePtr + col_out_degree.get()->size, static_cast<int>(-1), thrust::maximum<int>());
      return max_out_degree;
}

typedef struct Katz_Usecase_t {
  std::string matrix_file;
  std::string result_file;
  Katz_Usecase_t(const std::string& a, const std::string& b) {
    // assume relative paths are relative to RAPIDS_DATASET_ROOT_DIR
    const std::string& rapidsDatasetRootDir = get_rapids_dataset_root_dir();
    if ((a != "") && (a[0] != '/')) {
      matrix_file = rapidsDatasetRootDir + "/" + a;
    } else {
      matrix_file = a;
    }
    if ((b != "") && (b[0] != '/')) {
      result_file = rapidsDatasetRootDir + "/" + b;
    } else {
      result_file = b;
    }
  }
  Katz_Usecase_t& operator=(const Katz_Usecase_t& rhs) {
    matrix_file = rhs.matrix_file;
    result_file = rhs.result_file;
    return *this;
  }
} Katz_Usecase;

class Tests_Katz : public ::testing::TestWithParam<Katz_Usecase> {
 public:
  Tests_Katz() {}
  static void SetupTestCase() {}
  static void TearDownTestCase() {}
  virtual void SetUp() {}
  virtual void TearDown() {}

  void run_current_test(const Katz_Usecase& param) {
       Graph_ptr G{new cugraph::Graph, Graph_deleter};
       gdf_column_ptr col_src, col_dest, col_katz_centrality;

       FILE* fpin = fopen(param.matrix_file.c_str(),"r");
       ASSERT_NE(fpin, nullptr) << "fopen (" << param.matrix_file << ") failure.";

       std::ifstream fs_result(param.result_file);
       ASSERT_EQ(fs_result.is_open(), true) << "file open (" << param.result_file << ") failure.";

       int m, k;
       int nnz;
       MM_typecode mc;
       ASSERT_EQ(mm_properties<int>(fpin, 1, &mc, &m, &k, &nnz),0) << "could not read Matrix Market file properties"<< "\n";
       ASSERT_TRUE(mm_is_matrix(mc));
       ASSERT_TRUE(mm_is_coordinate(mc));
       ASSERT_FALSE(mm_is_complex(mc));
       ASSERT_FALSE(mm_is_skew(mc));

       // Allocate memory on host
       std::vector<int> cooRowInd(nnz), cooColInd(nnz);
       std::vector<int> cooVal(nnz);
       std::vector<double> katz_centrality(m);

       // Read
       ASSERT_EQ( (mm_to_coo<int,int>(fpin, 1, nnz, &cooRowInd[0], &cooColInd[0], &cooVal[0], NULL)) , 0)<< "could not read matrix data"<< "\n";
       ASSERT_EQ(fclose(fpin),0);

      // gdf columns
      col_src = create_gdf_column(cooRowInd);
      col_dest = create_gdf_column(cooColInd);
      col_katz_centrality = create_gdf_column(katz_centrality);

      cugraph::edge_list_view(G.get(), col_src.get(), col_dest.get(), nullptr);
      int max_out_degree = getMaxDegree(G.get());
      double alpha = 1/(static_cast<double>(max_out_degree) + 1);

      cugraph::katz_centrality(G.get(), col_katz_centrality.get(), alpha, 100, 1e-6, false, true);

      std::vector<int> top10CUGraph = getTopKIds(std::move(col_katz_centrality));
      std::vector<int> top10Golden  = getGoldenTopKIds(fs_result);

      EXPECT_THAT(top10CUGraph, ::testing::ContainerEq(top10Golden));
  }

};

// --gtest_filter=*simple_test*
INSTANTIATE_TEST_CASE_P(simple_test, Tests_Katz,
                        ::testing::Values(  Katz_Usecase("test/datasets/karate.mtx",      "ref/katz/karate.csv"    )
                                           ,Katz_Usecase("test/datasets/netscience.mtx",  "ref/katz/netscience.csv")
                                           ,Katz_Usecase("test/datasets/polbooks.mtx",    "ref/katz/polbooks.csv"  )
                                           ,Katz_Usecase("test/datasets/dolphins.mtx",    "ref/katz/dolphins.csv"  )
                                         )
                       );

TEST_P(Tests_Katz, Check) {
    run_current_test(GetParam());
}

int main( int argc, char** argv )
{
    rmmInitialize(nullptr);
    testing::InitGoogleTest(&argc,argv);
    int rc = RUN_ALL_TESTS();
    rmmFinalize();
    return rc;
}
